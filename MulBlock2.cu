#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <windows.h>
#include <hip/hip_runtime.h>

__host__ void initial(void);
__global__ void MatMul(int *dev_A,int *dev_B,int 

*dev_C,int M1,int N1,int M2,int N2);

const int blocksize=3;
int M1,N1,M2,N2;
int *A,*B,*C;
int *dev_A,*dev_B,*dev_C;
//clock_t start,finish;
double Time;
int main(){

	//printf("Enter blocksize:");
	//scanf("%d",&blocksize);
	M1=10*blocksize;
	N1=10*blocksize;
	M2=10*blocksize;
	N2=20*blocksize;

	initial();
	//printf("%d %d %d %d",M1,N1,M2,N2);
	int blocksPerGrid=blocksize*blocksize;
	int threadsPerBlock=blocksize;
	//start=clock();
	LARGE_INTEGER nFreq;  
    LARGE_INTEGER nBeginTime;  
    LARGE_INTEGER nEndTime;  
	QueryPerformanceFrequency(&nFreq);  
	QueryPerformanceCounter(&nBeginTime);
	//printf("a:%d,b:%d\n",blocksPerGrid,threadsPerBlock);
	//dim3 Dg(blocksPerGrid,threadsPerBlock,1);
	MatMul<<<blocksPerGrid,threadsPerBlock>>>(dev_A,dev_B,dev_C,M1,N1,M2,N2);
	QueryPerformanceCounter(&nEndTime);  
    Time=(double)(nEndTime.QuadPart-nBeginTime.QuadPart)/(double)nFreq.QuadPart; 
	//finish=clock();
	hipMemcpy((int*)C,(int*)dev_C,M1*N2*sizeof(int),hipMemcpyDeviceToHost);
	
	int i,j,index;
	for(i=0;i<M1;i++){
		for(j=0;j<N2;j++){
			index=i*N2+j;
			printf("%d ",C[index]);
		}
		printf("\n");
	}
	
	//Time=(double)(finish-start)/CLOCKS_PER_SEC;
	printf("Time:%lf\n",Time);
	system("pause");

    return 0;
}

__host__ void initial(void){

	int i,temp;
	A=(int*)malloc(M1*N1*sizeof(int));
	B=(int*)malloc(M2*N2*sizeof(int));
	C=(int*)malloc(M1*N2*sizeof(int));
	temp=M1*N1;
	for(i=0;i<temp;i++){
		A[i]=i;
	}
	temp=M2*N2;
	for(i=0;i<temp;i++){
		B[i]=i;
	}
	temp=M1*N2;
	for(i=0;i<temp;i++){
		C[i]=0;
	}
	hipMalloc((int**)&dev_A, M1*N1*sizeof(int)) 

;
    hipMalloc((int**)&dev_B, M2*N2*sizeof(int)) ;
    hipMalloc((int**)&dev_C, M1*N2*sizeof(int)) ;
	
	hipMemcpy((int*)dev_A,(int*)A,M1*N1*sizeof(int*),hipMemcpyHostToDevice);
	hipMemcpy((int*)dev_B,(int*)B,M2*N2*sizeof(int*),hipMemcpyHostToDevice);
	hipMemcpy((int*)dev_C,(int*)C,M1*N2*sizeof(int*),hipMemcpyHostToDevice);
}

__global__ void MatMul(int *dev_A,int *dev_B,int *dev_C,int M1,int N1,int M2,int N2){
	
	int x,y,m,index;
	__shared__ int cache[blocksize][10][20];
	//printf("yes\n");
	x=blockIdx.x;
	y=x%blocksize;
	x=x/blocksize;
	m=threadIdx.x;
	int i,j,k;
	int index_a,index_b;
	int ii,jj;
	//ii=10*(x+1);
	//jj=20*(y+1);
	for(i=0;i<10;i++){
		for(j=0;j<20;j++){
			cache[m][i][j]=0;
			ii=10*x+i;
			jj=20*y+j;
			//index=jj+ii*N2;
			for(k=10*m;k<10*(m+1);k++){
				index_a=ii*N1+k;
				index_b=k*N2+jj;
				cache[m][i][j]+=dev_A[index_a]*dev_B[index_b];
			}
		}
	}
	
	__syncthreads();
	if(m==0){
		for(i=1;i<blocksize;i++){
			for(j=0;j<10;j++){
				for(k=0;k<20;k++){
					cache[0][j][k]+=cache[i][j][k];
				}
			}
		}
		for(i=0;i<10;i++){
			for(j=0;j<20;j++){
				ii=10*x+i;
				jj=20*y+j;
				index=jj+ii*N2;
				dev_C[index]=cache[0][i][j];
			}
		}

	}
}
