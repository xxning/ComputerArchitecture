#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <windows.h>
#include <hip/hip_runtime.h>

__host__ void initial(void);
__global__ void MatMul(int *dev_A,int *dev_B,int *dev_C,int M1,int N1,int M2,int N2,int blocksize);

int blocksize;
int M1,N1,M2,N2;
int *A,*B,*C;
int *dev_A,*dev_B,*dev_C;
//clock_t start,finish;
double Time;
int main(){

	printf("Enter blocksize:");
	scanf("%d",&blocksize);
	M1=10*blocksize;
	N1=10*blocksize;
	M2=10*blocksize;
	N2=20*blocksize;

	initial();
	//printf("%d %d %d %d",M1,N1,M2,N2);
	int blocksPerGrid=blocksize;
	int threadsPerBlock=blocksize;
	//start=clock();
	LARGE_INTEGER nFreq;  
    LARGE_INTEGER nBeginTime;  
    LARGE_INTEGER nEndTime;  
	QueryPerformanceFrequency(&nFreq);  
	QueryPerformanceCounter(&nBeginTime);
	//printf("a:%d,b:%d\n",blocksPerGrid,threadsPerBlock);
	//dim3 Dg(blocksPerGrid,threadsPerBlock,1);
	MatMul<<<blocksPerGrid,threadsPerBlock>>>(dev_A,dev_B,dev_C,M1,N1,M2,N2,blocksize);
	QueryPerformanceCounter(&nEndTime);  
    Time=(double)(nEndTime.QuadPart-nBeginTime.QuadPart)/(double)nFreq.QuadPart; 
	//finish=clock();
	hipMemcpy((int*)C,(int*)dev_C,M1*N2*sizeof(int),hipMemcpyDeviceToHost);
	/*
	int i,j,index;
	for(i=0;i<M1;i++){
		for(j=0;j<N2;j++){
			index=i*N2+j;
			printf("%d ",C[index]);
		}
		printf("\n");
	}
	*/
	//Time=(double)(finish-start)/CLOCKS_PER_SEC;
	printf("Time:%lf\n",Time);
	system("pause");

    return 0;
}

__host__ void initial(void){

	int i,temp;
	A=(int*)malloc(M1*N1*sizeof(int));
	B=(int*)malloc(M2*N2*sizeof(int));
	C=(int*)malloc(M1*N2*sizeof(int));
	temp=M1*N1;
	for(i=0;i<temp;i++){
		A[i]=i;
	}
	temp=M2*N2;
	for(i=0;i<temp;i++){
		B[i]=i;
	}
	temp=M1*N2;
	for(i=0;i<temp;i++){
		C[i]=0;
	}
	hipMalloc((int**)&dev_A, M1*N1*sizeof(int)) ;
    hipMalloc((int**)&dev_B, M2*N2*sizeof(int)) ;
    hipMalloc((int**)&dev_C, M1*N2*sizeof(int)) ;
	
	hipMemcpy((int*)dev_A,(int*)A, M1*N1*sizeof(int*),hipMemcpyHostToDevice);
	hipMemcpy((int*)dev_B,(int*)B, M2*N2*sizeof(int*),hipMemcpyHostToDevice);
	hipMemcpy((int*)dev_C,(int*)C, M1*N2*sizeof(int*),hipMemcpyHostToDevice);
}

__global__ void MatMul(int *dev_A,int *dev_B,int *dev_C,int M1,int N1,int M2,int N2,int blocksize){
	
	int x,y,index;
	//printf("yes\n");
	x=blockIdx.x;
	y=threadIdx.x;
	int i,j,k;
	int index_a,index_b;
	int ii,jj;
	ii=10*(x+1);
	jj=20*(y+1);
	for(i=10*x;i<ii;i++){
		for(j=20*y;j<ii;j++){
			index=j+i*N2;
			dev_C[index]=0;
			for(k=0;k<N1;k++){	
				index_a=i*N1+k;
				index_b=k*N2+j;
				dev_C[index]+=dev_A[index_a]*dev_B[index_b];
			}
		}
	}
}
